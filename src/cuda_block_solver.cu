#include "hip/hip_runtime.h"
/*
Copyright 2020 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "cuda_block_solver.h"

#include <algorithm>
#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/gather.h>

namespace cuba
{
namespace gpu
{

////////////////////////////////////////////////////////////////////////////////////
// Type alias
////////////////////////////////////////////////////////////////////////////////////

template <int N>
using Vecxd = Vec<Scalar, N>;

template <int N>
using GpuVecxd = GpuVec<Vecxd<N>>;

using PxPBlockPtr = BlockPtr<Scalar, PDIM, PDIM>;
using LxLBlockPtr = BlockPtr<Scalar, LDIM, LDIM>;
using PxLBlockPtr = BlockPtr<Scalar, PDIM, LDIM>;
using Px1BlockPtr = BlockPtr<Scalar, PDIM, 1>;
using Lx1BlockPtr = BlockPtr<Scalar, LDIM, 1>;

////////////////////////////////////////////////////////////////////////////////////
// Constants
////////////////////////////////////////////////////////////////////////////////////
constexpr int BLOCK_ACTIVE_ERRORS = 512;
constexpr int BLOCK_MAX_DIAGONAL = 512;
constexpr int BLOCK_COMPUTE_SCALE = 512;

////////////////////////////////////////////////////////////////////////////////////
// Type definitions
////////////////////////////////////////////////////////////////////////////////////
struct LessRowId
{
	__device__ bool operator()(const Vec3i& lhs, const Vec3i& rhs) const
	{
		if (lhs[0] == rhs[0])
			return lhs[1] < rhs[1];
		return lhs[0] < rhs[0];
	}
};

struct LessColId
{
	__device__ bool operator()(const Vec3i& lhs, const Vec3i& rhs) const
	{
		if (lhs[1] == rhs[1])
			return lhs[0] < rhs[0];
		return lhs[1] < rhs[1];
	}
};

template <typename T, int ROWS, int COLS>
struct MatView
{
	__device__ inline T& operator()(int i, int j) { return data[j * ROWS + i]; }
	__device__ inline MatView(T* data) : data(data) {}
	T* data;
};

template <typename T, int ROWS, int COLS>
struct ConstMatView
{
	__device__ inline T operator()(int i, int j) const { return data[j * ROWS + i]; }
	__device__ inline ConstMatView(const T* data) : data(data) {}
	const T* data;
};

template <typename T, int ROWS, int COLS>
struct Matx
{
	using View = MatView<T, ROWS, COLS>;
	using ConstView = ConstMatView<T, ROWS, COLS>;
	__device__ inline T& operator()(int i, int j) { return data[j * ROWS + i]; }
	__device__ inline T operator()(int i, int j) const { return data[j * ROWS + i]; }
	__device__ inline operator View() { return View(data); }
	__device__ inline operator ConstView() const { return ConstView(data); }
	T data[ROWS * COLS];
};

using MatView2x3d = MatView<Scalar, 2, 3>;
using MatView2x6d = MatView<Scalar, 2, 6>;
using MatView3x1d = MatView<Scalar, 3, 1>;
using MatView3x3d = MatView<Scalar, 3, 3>;
using MatView3x6d = MatView<Scalar, 3, 6>;
using ConstMatView3x1d = ConstMatView<Scalar, 3, 1>;
using ConstMatView3x3d = ConstMatView<Scalar, 3, 3>;
using ConstMatView6x6d = ConstMatView<Scalar, 6, 6>;
using ConstMatView6x1d = ConstMatView<Scalar, 6, 1>;

struct CameraParamView
{
	__device__ inline CameraParamView(const Scalar* data) : data(data) {}
	__device__ inline CameraParamView(const Vec5d& camera) : data(camera.data) {}
	__device__ inline Scalar fx() const { return data[0]; }
	__device__ inline Scalar fy() const { return data[1]; }
	__device__ inline Scalar cx() const { return data[2]; }
	__device__ inline Scalar cy() const { return data[3]; }
	__device__ inline Scalar bf() const { return data[4]; }

	const Scalar* data;
};

////////////////////////////////////////////////////////////////////////////////////
// Host functions
////////////////////////////////////////////////////////////////////////////////////
static int divUp(int total, int grain)
{
	return (total + grain - 1) / grain;
}

////////////////////////////////////////////////////////////////////////////////////
// Device functions (template matrix and verctor operation)
////////////////////////////////////////////////////////////////////////////////////

// assignment operations
using AssignOP = void(*)(Scalar*, Scalar);
__device__ inline void ASSIGN(Scalar* address, Scalar value) { *address = value; }
__device__ inline void ACCUM(Scalar* address, Scalar value) { *address += value; }
__device__ inline void DEACCUM(Scalar* address, Scalar value) { *address -= value; }
__device__ inline void ACCUM_ATOMIC(Scalar* address, Scalar value) { atomicAdd(address, value); }
__device__ inline void DEACCUM_ATOMIC(Scalar* address, Scalar value) { atomicAdd(address, -value); }

// recursive dot product for inline expansion
template <int N>
__device__ inline Scalar dot_(const Scalar* a, const Scalar* b)
{
	return dot_<N - 1>(a, b) + a[N - 1] * b[N - 1];
}

template <>
__device__ inline Scalar dot_<1>(const Scalar* a, const Scalar* b) { return a[0] * b[0]; }

// recursive dot product for inline expansion (strided access pattern)
template <int N, int S1, int S2>
__device__ inline Scalar dot_stride_(const Scalar* a, const Scalar* b)
{
	static_assert(S1 == PDIM || S1 == LDIM, "S1 must be PDIM or LDIM");
	static_assert(S2 == 1 || S2 == PDIM || S2 == LDIM, "S2 must be 1 or PDIM or LDIM");
	return dot_stride_<N - 1, S1, S2>(a, b) + a[S1 * (N - 1)] * b[S2 * (N - 1)];
}

template <>
__device__ inline Scalar dot_stride_<1, PDIM, 1>(const Scalar* a, const Scalar* b) { return a[0] * b[0]; }
template <>
__device__ inline Scalar dot_stride_<1, LDIM, 1>(const Scalar* a, const Scalar* b) { return a[0] * b[0]; }
template <>
__device__ inline Scalar dot_stride_<1, PDIM, PDIM>(const Scalar* a, const Scalar* b) { return a[0] * b[0]; }
template <>
__device__ inline Scalar dot_stride_<1, LDIM, LDIM>(const Scalar* a, const Scalar* b) { return a[0] * b[0]; }

// matrix(tansposed)-vector product: b = AT*x
template <int M, int N, AssignOP OP = ASSIGN>
__device__ inline void MatTMulVec(const Scalar* A, const Scalar* x, Scalar* b, Scalar omega)
{
#pragma unroll
	for (int i = 0; i < M; i++)
		OP(b + i, omega * dot_<N>(A + i * N, x));
}

// matrix(tansposed)-matrix product: C = AT*B
template <int L, int M, int N, AssignOP OP = ASSIGN>
__device__ inline void MatTMulMat(const Scalar* A, const Scalar* B, Scalar* C, Scalar omega)
{
#pragma unroll
	for (int i = 0; i < N; i++)
		MatTMulVec<L, M, OP>(A, B + i * M, C + i * L, omega);
}

// matrix-vector product: b = A*x
template <int M, int N, int S = 1, AssignOP OP = ASSIGN>
__device__ inline void MatMulVec(const Scalar* A, const Scalar* x, Scalar* b)
{
#pragma unroll
	for (int i = 0; i < M; i++)
		OP(b + i, dot_stride_<N, M, S>(A + i, x));
}

// matrix-matrix product: C = A*B
template <int L, int M, int N, AssignOP OP = ASSIGN>
__device__ inline void MatMulMat(const Scalar* A, const Scalar* B, Scalar* C)
{
#pragma unroll
	for (int i = 0; i < N; i++)
		MatMulVec<L, M, 1, OP>(A, B + i * M, C + i * L);
}

// matrix-matrix(tansposed) product: C = A*BT
template <int L, int M, int N, AssignOP OP = ASSIGN>
__device__ inline void MatMulMatT(const Scalar* A, const Scalar* B, Scalar* C)
{
#pragma unroll
	for (int i = 0; i < N; i++)
		MatMulVec<L, M, N, OP>(A, B + i, C + i * L);
}

// squared L2 norm
template <int N>
__device__ inline Scalar squaredNorm(const Scalar* x) { return dot_<N>(x, x); }
template <int N>
__device__ inline Scalar squaredNorm(const Vecxd<N>& x) { return squaredNorm<N>(x.data); }

// L2 norm
template <int N>
__device__ inline Scalar norm(const Scalar* x) { return sqrt(squaredNorm<N>(x)); }
template <int N>
__device__ inline Scalar norm(const Vecxd<N>& x) { return norm<N>(x.data); }

////////////////////////////////////////////////////////////////////////////////////
// Device functions
////////////////////////////////////////////////////////////////////////////////////
__device__ static inline void cross(const Vec4d& a, const Vec3d& b, Vec3d& c)
{
	c[0] = a[1] * b[2] - a[2] * b[1];
	c[1] = a[2] * b[0] - a[0] * b[2];
	c[2] = a[0] * b[1] - a[1] * b[0];
}

__device__ inline void rotate(const Vec4d& q, const Vec3d& Xw, Vec3d& Xc)
{
	Vec3d tmp1, tmp2;

	cross(q, Xw, tmp1);

	tmp1[0] += tmp1[0];
	tmp1[1] += tmp1[1];
	tmp1[2] += tmp1[2];

	cross(q, tmp1, tmp2);

	Xc[0] = Xw[0] + q[3] * tmp1[0] + tmp2[0];
	Xc[1] = Xw[1] + q[3] * tmp1[1] + tmp2[1];
	Xc[2] = Xw[2] + q[3] * tmp1[2] + tmp2[2];
}

__device__ inline void projectW2C(const Vec4d& q, const Vec3d& t, const Vec3d& Xw, Vec3d& Xc)
{
	rotate(q, Xw, Xc);
	Xc[0] += t[0];
	Xc[1] += t[1];
	Xc[2] += t[2];
}

template <int MDIM>
__device__ inline void projectC2I(const Vec3d& Xc, Vecxd<MDIM>& p, CameraParamView camera)
{
}

template <>
__device__ inline void projectC2I<2>(const Vec3d& Xc, Vec2d& p, CameraParamView camera)
{
	const Scalar invZ = 1 / Xc[2];
	p[0] = camera.fx() * invZ * Xc[0] + camera.cx();
	p[1] = camera.fy() * invZ * Xc[1] + camera.cy();
}

template <>
__device__ inline void projectC2I<3>(const Vec3d& Xc, Vec3d& p, CameraParamView camera)
{
	const Scalar invZ = 1 / Xc[2];
	p[0] = camera.fx() * invZ * Xc[0] + camera.cx();
	p[1] = camera.fy() * invZ * Xc[1] + camera.cy();
	p[2] = p[0] - camera.bf() * invZ;
}

__device__ inline void quaternionToRotationMatrix(const Vec4d& q, MatView3x3d R)
{
	const Scalar x = q[0];
	const Scalar y = q[1];
	const Scalar z = q[2];
	const Scalar w = q[3];

	const Scalar tx = 2 * x;
	const Scalar ty = 2 * y;
	const Scalar tz = 2 * z;
	const Scalar twx = tx * w;
	const Scalar twy = ty * w;
	const Scalar twz = tz * w;
	const Scalar txx = tx * x;
	const Scalar txy = ty * x;
	const Scalar txz = tz * x;
	const Scalar tyy = ty * y;
	const Scalar tyz = tz * y;
	const Scalar tzz = tz * z;

	R(0, 0) = 1 - (tyy + tzz);
	R(0, 1) = txy - twz;
	R(0, 2) = txz + twy;
	R(1, 0) = txy + twz;
	R(1, 1) = 1 - (txx + tzz);
	R(1, 2) = tyz - twx;
	R(2, 0) = txz - twy;
	R(2, 1) = tyz + twx;
	R(2, 2) = 1 - (txx + tyy);
}

template <int MDIM>
__device__ void computeJacobians(const Vec3d& Xc, const Vec4d& q,
	MatView<Scalar, MDIM, PDIM> JP, MatView<Scalar, MDIM, LDIM> JL, CameraParamView camera)
{
}

template <>
__device__ void computeJacobians<2>(const Vec3d& Xc, const Vec4d& q, MatView2x6d JP, MatView2x3d JL, CameraParamView camera)
{
	const Scalar X = Xc[0];
	const Scalar Y = Xc[1];
	const Scalar Z = Xc[2];
	const Scalar invZ = 1 / Z;
	const Scalar x = invZ * X;
	const Scalar y = invZ * Y;
	const Scalar fu = camera.fx();
	const Scalar fv = camera.fy();
	const Scalar fu_invZ = fu * invZ;
	const Scalar fv_invZ = fv * invZ;

	Matx<Scalar, 3, 3> R;
	quaternionToRotationMatrix(q, R);

	JL(0, 0) = -fu_invZ * (R(0, 0) - x * R(2, 0));
	JL(0, 1) = -fu_invZ * (R(0, 1) - x * R(2, 1));
	JL(0, 2) = -fu_invZ * (R(0, 2) - x * R(2, 2));
	JL(1, 0) = -fv_invZ * (R(1, 0) - y * R(2, 0));
	JL(1, 1) = -fv_invZ * (R(1, 1) - y * R(2, 1));
	JL(1, 2) = -fv_invZ * (R(1, 2) - y * R(2, 2));

	JP(0, 0) = +fu * x * y;
	JP(0, 1) = -fu * (1 + x * x);
	JP(0, 2) = +fu * y;
	JP(0, 3) = -fu_invZ;
	JP(0, 4) = 0;
	JP(0, 5) = +fu_invZ * x;

	JP(1, 0) = +fv * (1 + y * y);
	JP(1, 1) = -fv * x * y;
	JP(1, 2) = -fv * x;
	JP(1, 3) = 0;
	JP(1, 4) = -fv_invZ;
	JP(1, 5) = +fv_invZ * y;
}

template <>
__device__ void computeJacobians<3>(const Vec3d& Xc, const Vec4d& q, MatView3x6d JP, MatView3x3d JL, CameraParamView camera)
{
	const Scalar X = Xc[0];
	const Scalar Y = Xc[1];
	const Scalar Z = Xc[2];
	const Scalar invZ = 1 / Z;
	const Scalar invZZ = invZ * invZ;
	const Scalar fu = camera.fx();
	const Scalar fv = camera.fy();
	const Scalar bf = camera.bf();

	Matx<Scalar, 3, 3> R;
	quaternionToRotationMatrix(q, R);

	JL(0, 0) = -fu * R(0, 0) * invZ + fu * X * R(2, 0) * invZZ;
	JL(0, 1) = -fu * R(0, 1) * invZ + fu * X * R(2, 1) * invZZ;
	JL(0, 2) = -fu * R(0, 2) * invZ + fu * X * R(2, 2) * invZZ;

	JL(1, 0) = -fv * R(1, 0) * invZ + fv * Y * R(2, 0) * invZZ;
	JL(1, 1) = -fv * R(1, 1) * invZ + fv * Y * R(2, 1) * invZZ;
	JL(1, 2) = -fv * R(1, 2) * invZ + fv * Y * R(2, 2) * invZZ;

	JL(2, 0) = JL(0, 0) - bf * R(2, 0) * invZZ;
	JL(2, 1) = JL(0, 1) - bf * R(2, 1) * invZZ;
	JL(2, 2) = JL(0, 2) - bf * R(2, 2) * invZZ;

	JP(0, 0) = X * Y * invZZ * fu;
	JP(0, 1) = -(1 + (X * X * invZZ)) * fu;
	JP(0, 2) = Y * invZ * fu;
	JP(0, 3) = -1 * invZ * fu;
	JP(0, 4) = 0;
	JP(0, 5) = X * invZZ * fu;

	JP(1, 0) = (1 + Y * Y * invZZ) * fv;
	JP(1, 1) = -X * Y * invZZ * fv;
	JP(1, 2) = -X * invZ * fv;
	JP(1, 3) = 0;
	JP(1, 4) = -1 * invZ * fv;
	JP(1, 5) = Y * invZZ * fv;

	JP(2, 0) = JP(0, 0) - bf * Y * invZZ;
	JP(2, 1) = JP(0, 1) + bf * X * invZZ;
	JP(2, 2) = JP(0, 2);
	JP(2, 3) = JP(0, 3);
	JP(2, 4) = 0;
	JP(2, 5) = JP(0, 5) - bf * invZZ;
}

__device__ inline void Sym3x3Inv(ConstMatView3x3d A, MatView3x3d B)
{
	const Scalar A00 = A(0, 0);
	const Scalar A01 = A(0, 1);
	const Scalar A11 = A(1, 1);
	const Scalar A02 = A(2, 0);
	const Scalar A12 = A(1, 2);
	const Scalar A22 = A(2, 2);

	const Scalar det
		= A00 * A11 * A22
		+ A01 * A12 * A02
		+ A02 * A01 * A12
		- A00 * A12 * A12
		- A02 * A11 * A02
		- A01 * A01 * A22;

	const Scalar invDet = 1 / det;

	const Scalar B00 = invDet * (A11 * A22 - A12 * A12);
	const Scalar B01 = invDet * (A02 * A12 - A01 * A22);
	const Scalar B11 = invDet * (A00 * A22 - A02 * A02);
	const Scalar B02 = invDet * (A01 * A12 - A02 * A11);
	const Scalar B12 = invDet * (A02 * A01 - A00 * A12);
	const Scalar B22 = invDet * (A00 * A11 - A01 * A01);

	B(0, 0) = B00;
	B(0, 1) = B01;
	B(0, 2) = B02;
	B(1, 0) = B01;
	B(1, 1) = B11;
	B(1, 2) = B12;
	B(2, 0) = B02;
	B(2, 1) = B12;
	B(2, 2) = B22;
}

__device__ inline void skew1(Scalar x, Scalar y, Scalar z, MatView3x3d M)
{
	M(0, 0) = +0; M(0, 1) = -z; M(0, 2) = +y;
	M(1, 0) = +z; M(1, 1) = +0; M(1, 2) = -x;
	M(2, 0) = -y; M(2, 1) = +x; M(2, 2) = +0;
}

__device__ inline void skew2(Scalar x, Scalar y, Scalar z, MatView3x3d M)
{
	const Scalar xx = x * x;
	const Scalar yy = y * y;
	const Scalar zz = z * z;

	const Scalar xy = x * y;
	const Scalar yz = y * z;
	const Scalar zx = z * x;

	M(0, 0) = -yy - zz; M(0, 1) = +xy;      M(0, 2) = +zx;
	M(1, 0) = +xy;      M(1, 1) = -zz - xx; M(1, 2) = +yz;
	M(2, 0) = +zx;      M(2, 1) = +yz;      M(2, 2) = -xx - yy;
}

__device__ inline void addOmega(Scalar a1, ConstMatView3x3d O1, Scalar a2, ConstMatView3x3d O2,
	MatView3x3d R)
{
	R(0, 0) = 1 + a1 * O1(0, 0) + a2 * O2(0, 0);
	R(1, 0) = 0 + a1 * O1(1, 0) + a2 * O2(1, 0);
	R(2, 0) = 0 + a1 * O1(2, 0) + a2 * O2(2, 0);

	R(0, 1) = 0 + a1 * O1(0, 1) + a2 * O2(0, 1);
	R(1, 1) = 1 + a1 * O1(1, 1) + a2 * O2(1, 1);
	R(2, 1) = 0 + a1 * O1(2, 1) + a2 * O2(2, 1);

	R(0, 2) = 0 + a1 * O1(0, 2) + a2 * O2(0, 2);
	R(1, 2) = 0 + a1 * O1(1, 2) + a2 * O2(1, 2);
	R(2, 2) = 1 + a1 * O1(2, 2) + a2 * O2(2, 2);
}

__device__ inline void rotationMatrixToQuaternion(ConstMatView3x3d R, Vec4d& q)
{
	Scalar t = R(0, 0) + R(1, 1) + R(2, 2);
	if (t > 0)
	{
		t = sqrt(t + 1);
		q[3] = Scalar(0.5) * t;
		t = Scalar(0.5) / t;
		q[0] = (R(2, 1) - R(1, 2)) * t;
		q[1] = (R(0, 2) - R(2, 0)) * t;
		q[2] = (R(1, 0) - R(0, 1)) * t;
	}
	else
	{
		int i = 0;
		if (R(1, 1) > R(0, 0))
			i = 1;
		if (R(2, 2) > R(i, i))
			i = 2;
		int j = (i + 1) % 3;
		int k = (j + 1) % 3;

		t = sqrt(R(i, i) - R(j, j) - R(k, k) + 1);
		q[i] = Scalar(0.5) * t;
		t = Scalar(0.5) / t;
		q[3] = (R(k, j) - R(j, k)) * t;
		q[j] = (R(j, i) + R(i, j)) * t;
		q[k] = (R(k, i) + R(i, k)) * t;
	}
}

__device__ inline void multiplyQuaternion(const Vec4d& a, const Vec4d& b, Vec4d& c)
{
	c[3] = a[3] * b[3] - a[0] * b[0] - a[1] * b[1] - a[2] * b[2];
	c[0] = a[3] * b[0] + a[0] * b[3] + a[1] * b[2] - a[2] * b[1];
	c[1] = a[3] * b[1] + a[1] * b[3] + a[2] * b[0] - a[0] * b[2];
	c[2] = a[3] * b[2] + a[2] * b[3] + a[0] * b[1] - a[1] * b[0];
}

__device__ inline void normalizeQuaternion(const Vec4d& a, Vec4d& b)
{
	Scalar invn = 1 / norm(a);
	if (a[3] < 0)
		invn = -invn;

	for (int i = 0; i < 4; i++)
		b[i] = invn * a[i];
}

__device__ inline Scalar pow2(Scalar x)
{
	return x * x;
}

__device__ inline Scalar pow3(Scalar x)
{
	return x * x * x;
}

__device__ inline void updateExp(const Scalar* update, Vec4d& q, Vec3d& t)
{
	Vec3d omega(update);
	Vec3d upsilon(update + 3);

	const Scalar theta = norm(omega);

	Matx<Scalar, 3, 3> O1, O2;
	skew1(omega[0], omega[1], omega[2], O1);
	skew2(omega[0], omega[1], omega[2], O2);

	Scalar R[9], V[9];
	if (theta < Scalar(0.00001))
	{
		addOmega(Scalar(1.0), O1, Scalar(0.5), O2, R);
		addOmega(Scalar(0.5), O1, Scalar(1)/6, O2, V);
	}
	else
	{
		const Scalar a1 = sin(theta) / theta;
		const Scalar a2 = (1 - cos(theta)) / (theta * theta);
		const Scalar a3 = (theta - sin(theta)) / pow3(theta);
		addOmega(a1, O1, a2, O2, R);
		addOmega(a2, O1, a3, O2, V);
	}

	rotationMatrixToQuaternion(R, q);
	MatMulVec<3, 3>(V, upsilon.data, t.data);
}

__device__ inline void updatePose(const Vec4d& q1, const Vec3d& t1, Vec4d& q2, Vec3d& t2)
{
	Vec3d u;
	rotate(q1, t2, u);

	for (int i = 0; i < 3; i++)
		t2[i] = t1[i] + u[i];

	Vec4d r;
	multiplyQuaternion(q1, q2, r);
	normalizeQuaternion(r, q2);
}

template <int N>
__device__ inline void copy(const Scalar* src, Scalar* dst)
{
	for (int i = 0; i < N; i++)
		dst[i] = src[i];
}

__device__ inline Vec3i makeVec3i(int i, int j, int k)
{
	Vec3i  vec;
	vec[0] = i;
	vec[1] = j;
	vec[2] = k;
	return vec;
}

__device__ inline  void solveSym3x3(const Scalar* H, const Scalar* b, Scalar* x)
{
	Scalar invH[LDIM * LDIM];
	Sym3x3Inv(H, invH);
	MatMulVec<3, 3>(invH, b, x);
}

__device__ inline  void solveSym6x6(const Scalar* _H, const Scalar* _b, Scalar* _x)
{
	using Mat3x3d = Matx<Scalar, 3, 3>;
	using Mat3x1d = Matx<Scalar, 3, 1>;
	using Mat6x1d = Matx<Scalar, 6, 1>;

	ConstMatView6x6d H(_H);
	ConstMatView6x1d b(_b);
	ConstMatView3x1d bp(b.data);
	ConstMatView3x1d bl(b.data + 3);

	Scalar buf1[LDIM * LDIM], buf2[LDIM * LDIM], buf3[LDIM * LDIM], buf4[LDIM];

	MatView3x3d Hpl(buf1);
	MatView3x3d Hll(buf2);
	for (int j = 0; j < 3; j++) for (int i = 0; i < 3; i++) Hpl(i, j) = H(i + 0, j + 3);
	for (int j = 0; j < 3; j++) for (int i = 0; i < 3; i++) Hll(i, j) = H(i + 3, j + 3);

	Mat6x1d x;
	MatView3x1d xp(x.data);
	MatView3x1d xl(x.data + 3);
	MatView3x3d invHll(buf3), Hpl_invHll(buf2);

	// Hsc = Hpp - Hpl*Hll^-1*HplT
	Mat3x3d Hsc;
	for (int j = 0; j < 3; j++) for (int i = 0; i < 3; i++) Hsc(i, j) = H(i, j);
	Sym3x3Inv(Hll.data, invHll.data);
	MatMulMat<3, 3, 3>(Hpl.data, invHll.data, Hpl_invHll.data);
	MatMulMatT<3, 3, 3, DEACCUM>(Hpl_invHll.data, Hpl.data, Hsc.data);

	// bsc = -bp + Hpl*Hll^-1*bl
	MatView3x1d bsc(buf4);
	copy<3>(bp.data, bsc.data);
	MatMulVec<3, 3, 1, DEACCUM>(Hpl_invHll.data, bl.data, bsc.data);

	// Hsc*Δxp = bsc
	MatView3x3d invHsc(buf2);
	Sym3x3Inv(Hsc, invHsc);
	MatMulVec<3, 3>(invHsc.data, bsc.data, xp.data);

	// Hll*Δxl = -bl - HplT*Δxp
	MatView3x1d cl(buf4);
	copy<3>(bl.data, cl.data);
	MatTMulVec<3, 3, DEACCUM>(Hpl.data, xp.data, cl.data, 1);
	MatMulVec<3, 3>(invHll.data, cl.data, xl.data);

	copy<6>(x.data, _x);
}

////////////////////////////////////////////////////////////////////////////////////
// Robust kernels
////////////////////////////////////////////////////////////////////////////////////
enum RobustKernelType
{
	NONE  = 0,
	HUBER = 1,
	TUKEY = 2,
};

template <int TYPE>
struct RobustKernelFunc
{
	__device__ inline RobustKernelFunc(Scalar delta) {}
	__device__ inline Scalar robustify(Scalar x) const { return x; }
	__device__ inline Scalar derivative(Scalar x) const { return 1; }
};

template <>
struct RobustKernelFunc<RobustKernelType::NONE>
{
	__device__ inline RobustKernelFunc(Scalar delta) {}
	__device__ inline Scalar robustify(Scalar x) const { return x; }
	__device__ inline Scalar derivative(Scalar x) const { return 1; }
};

template <>
struct RobustKernelFunc<RobustKernelType::HUBER>
{
	__device__ inline RobustKernelFunc(Scalar delta) : delta(delta), deltaSq(delta * delta) {}

	__device__ inline Scalar robustify(Scalar x) const
	{
		return x <= deltaSq ? x : (2 * sqrt(x) * delta - deltaSq);
	}

	__device__ inline Scalar derivative(Scalar x) const
	{
		return x <= deltaSq ? 1 : (delta / sqrt(x));
	}

	Scalar delta, deltaSq;
};

template <>
struct RobustKernelFunc<RobustKernelType::TUKEY>
{
	__device__ inline RobustKernelFunc(Scalar delta) : delta(delta), deltaSq(delta * delta) {}

	__device__ inline Scalar robustify(Scalar x) const
	{
		const Scalar maxv = (Scalar(1) / 3) * deltaSq;
		return x <= deltaSq ? maxv * (1 - pow3(1 - x / deltaSq)) : maxv;
	}

	__device__ inline Scalar derivative(Scalar x) const
	{
		return x <= deltaSq ? pow2(1 - x / deltaSq) : 0;
	}

	Scalar delta, deltaSq;
};

////////////////////////////////////////////////////////////////////////////////////
// Kernel functions
////////////////////////////////////////////////////////////////////////////////////
template <int MDIM, int RK_TYPE>
__global__ void computeActiveErrorsKernel(int nedges, const Vec4d* qs, const Vec3d* ts, const Vec5d* cameras,
	const Vec3d* Xws, const Vecxd<MDIM>* measurements, const Scalar* omegas, const Vec2i* edge2PL,
	RobustKernelFunc<RK_TYPE> robustKernel, Vecxd<MDIM>* errors, Vec3d* Xcs, Scalar* chi)
{
	using Vecmd = Vecxd<MDIM>;

	const int sharedIdx = threadIdx.x;
	__shared__ Scalar cache[BLOCK_ACTIVE_ERRORS];

	Scalar sumchi = 0;
	for (int iE = blockIdx.x * blockDim.x + threadIdx.x; iE < nedges; iE += gridDim.x * blockDim.x)
	{
		const Vec2i index = edge2PL[iE];
		const int iP = index[0];
		const int iL = index[1];

		const Vec4d& q = qs[iP];
		const Vec3d& t = ts[iP];
		const Vec5d& camera = cameras[iP];
		const Vec3d& Xw = Xws[iL];
		const Vecmd& measurement = measurements[iE];

		// project world to camera
		Vec3d Xc;
		projectW2C(q, t, Xw, Xc);

		// project camera to image
		Vecmd proj;
		projectC2I(Xc, proj, camera);

		// compute residual
		Vecmd error;
		for (int i = 0; i < MDIM; i++)
			error[i] = proj[i] - measurement[i];

		errors[iE] = error;
		Xcs[iE] = Xc;

		sumchi += robustKernel.robustify(omegas[iE] * squaredNorm(error));
	}

	cache[sharedIdx] = sumchi;
	__syncthreads();

	for (int stride = BLOCK_ACTIVE_ERRORS / 2; stride > 0; stride >>= 1)
	{
		if (sharedIdx < stride)
			cache[sharedIdx] += cache[sharedIdx + stride];
		__syncthreads();
	}

	if (sharedIdx == 0)
		atomicAdd(chi, cache[0]);
}

template <int MDIM, int RK_TYPE>
__global__ void constructQuadraticFormKernel(int nedges, const Vec3d* Xcs, const Vec4d* qs, const Vec5d* cameras, const Vecxd<MDIM>* errors,
	const Scalar* omegas, const Vec2i* edge2PL, const int* edge2Hpl, const uint8_t* flags, RobustKernelFunc<RK_TYPE> robustKernel,
	PxPBlockPtr Hpp, Px1BlockPtr bp, LxLBlockPtr Hll, Lx1BlockPtr bl, PxLBlockPtr Hpl)
{
	using Vecmd = Vecxd<MDIM>;

	const int iE = blockIdx.x * blockDim.x + threadIdx.x;
	if (iE >= nedges)
		return;

	const int iP = edge2PL[iE][0];
	const int iL = edge2PL[iE][1];
	const int flag = flags[iE];

	const Vec4d& q = qs[iP];
	const Vec5d& camera = cameras[iP];
	const Vec3d& Xc = Xcs[iE];
	const Vecmd& error = errors[iE];

	// Robust kernel derivative
	const Scalar e = squaredNorm(error) * omegas[iE];
	const Scalar rho1 = robustKernel.derivative(e);
	const Scalar omega = omegas[iE] * rho1;

	// compute Jacobians
	Scalar JP[MDIM * PDIM];
	Scalar JL[MDIM * LDIM];
	computeJacobians<MDIM>(Xc, q, JP, JL, camera);

	if (!(flag & EDGE_FLAG_FIXED_P))
	{
		// Hpp += = JPT*Ω*JP
		MatTMulMat<PDIM, MDIM, PDIM, ACCUM_ATOMIC>(JP, JP, Hpp.at(iP), omega);

		// bp += = JPT*Ω*r
		MatTMulVec<PDIM, MDIM, ACCUM_ATOMIC>(JP, error.data, bp.at(iP), omega);
	}
	if (!(flag & EDGE_FLAG_FIXED_L))
	{
		// Hll += = JLT*Ω*JL
		MatTMulMat<LDIM, MDIM, LDIM, ACCUM_ATOMIC>(JL, JL, Hll.at(iL), omega);

		// bl += = JLT*Ω*r
		MatTMulVec<LDIM, MDIM, ACCUM_ATOMIC>(JL, error.data, bl.at(iL), omega);
	}
	if (!flag)
	{
		// Hpl += = JPT*Ω*JL
		MatTMulMat<PDIM, MDIM, LDIM, ASSIGN>(JP, JL, Hpl.at(edge2Hpl[iE]), omega);
	}
}

template <int MDIM>
__global__ void computeChiSquaresKernel(int nedges, const Vec4d* qs, const Vec3d* ts, const Vec5d* cameras,
	const Vec3d* Xws, const Vecxd<MDIM>* measurements, const Scalar* omegas, const Vec2i* edge2PL, Scalar* chiSqs)
{
	using Vecmd = Vecxd<MDIM>;

	const int iE = blockIdx.x * blockDim.x + threadIdx.x;
	if (iE >= nedges)
		return;

	const Vec2i index = edge2PL[iE];
	const int iP = index[0];
	const int iL = index[1];

	const Vec4d& q = qs[iP];
	const Vec3d& t = ts[iP];
	const Vec5d& camera = cameras[iP];
	const Vec3d& Xw = Xws[iL];
	const Vecmd& measurement = measurements[iE];

	// project world to camera
	Vec3d Xc;
	projectW2C(q, t, Xw, Xc);

	// project camera to image
	Vecmd proj;
	projectC2I(Xc, proj, camera);

	// compute residual
	Vecmd error;
	for (int i = 0; i < MDIM; i++)
		error[i] = proj[i] - measurement[i];

	chiSqs[iE] = omegas[iE] * squaredNorm(error);
}

template <int DIM>
__global__ void maxDiagonalKernel(int size, const Scalar* D, Scalar* maxD)
{
	const int sharedIdx = threadIdx.x;
	__shared__ Scalar cache[BLOCK_MAX_DIAGONAL];

	Scalar maxVal = 0;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x)
	{
		const int j = i / DIM;
		const int k = i % DIM;
		const Scalar* ptrBlock = D + j * DIM * DIM;
		maxVal = max(maxVal, ptrBlock[k * DIM + k]);
	}

	cache[sharedIdx] = maxVal;
	__syncthreads();

	for (int stride = BLOCK_MAX_DIAGONAL / 2; stride > 0; stride >>= 1)
	{
		if (sharedIdx < stride)
			cache[sharedIdx] = max(cache[sharedIdx], cache[sharedIdx + stride]);
		__syncthreads();
	}

	if (sharedIdx == 0)
		maxD[blockIdx.x] = cache[0];
}

template <int DIM>
__global__ void addLambdaKernel(int size, Scalar* D, Scalar lambda, Scalar* backup)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	const int j = i / DIM;
	const int k = i % DIM;
	Scalar* ptrBlock = D + j * DIM * DIM;
	backup[i] = ptrBlock[k * DIM + k];
	ptrBlock[k * DIM + k] += lambda;
}

template <int DIM>
__global__ void restoreDiagonalKernel(int size, Scalar* D, const Scalar* backup)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	const int j = i / DIM;
	const int k = i % DIM;
	Scalar* ptrBlock = D + j * DIM * DIM;
	ptrBlock[k * DIM + k] = backup[i];
}

__global__ void computeBschureKernel(int cols, LxLBlockPtr Hll, LxLBlockPtr invHll,
	Lx1BlockPtr bl, PxLBlockPtr Hpl, const int* HplColPtr, const int* HplRowInd,
	Px1BlockPtr bsc, PxLBlockPtr Hpl_invHll)
{
	const int colId = blockIdx.x * blockDim.x + threadIdx.x;
	if (colId >= cols)
		return;

	Scalar iHll[LDIM * LDIM];
	Scalar Hpl_iHll[PDIM * LDIM];

	Sym3x3Inv(Hll.at(colId), iHll);
	copy<LDIM * LDIM>(iHll, invHll.at(colId));

	for (int i = HplColPtr[colId]; i < HplColPtr[colId + 1]; i++)
	{
		MatMulMat<6, 3, 3>(Hpl.at(i), iHll, Hpl_iHll);
		MatMulVec<6, 3, 1, DEACCUM_ATOMIC>(Hpl_iHll, bl.at(colId), bsc.at(HplRowInd[i]));
		copy<PDIM * LDIM>(Hpl_iHll, Hpl_invHll.at(i));
	}
}

__global__ void initializeHschurKernel(int rows, PxPBlockPtr Hpp, PxPBlockPtr Hsc, const int* HscRowPtr)
{
	const int rowId = blockIdx.x * blockDim.x + threadIdx.x;
	if (rowId >= rows)
		return;

	copy<PDIM * PDIM>(Hpp.at(rowId), Hsc.at(HscRowPtr[rowId]));
}

__global__ void computeHschureKernel(int size, const Vec3i* mulBlockIds,
	PxLBlockPtr Hpl_invHll, PxLBlockPtr Hpl, PxPBlockPtr Hschur)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= size)
		return;

	const Vec3i index = mulBlockIds[tid];
	Scalar A[PDIM * LDIM];
	Scalar B[PDIM * LDIM];
	copy<PDIM * LDIM>(Hpl_invHll.at(index[0]), A);
	copy<PDIM * LDIM>(Hpl.at(index[1]), B);
	MatMulMatT<6, 3, 6, DEACCUM_ATOMIC>(A, B, Hschur.at(index[2]));
}

__global__ void findHschureMulBlockIndicesKernel(int cols, const int* HplColPtr, const int* HplRowInd,
	const int* HscRowPtr, const int* HscColInd, Vec3i* mulBlockIds, int* nindices)
{
	const int colId = blockIdx.x * blockDim.x + threadIdx.x;
	if (colId >= cols)
		return;

	const int i0 = HplColPtr[colId];
	const int i1 = HplColPtr[colId + 1];
	for (int i = i0; i < i1; i++)
	{
		const int iP1 = HplRowInd[i];
		int k = HscRowPtr[iP1];
		for (int j = i; j < i1; j++)
		{
			const int iP2 = HplRowInd[j];
			while (HscColInd[k] < iP2) k++;
			const int pos = atomicAdd(nindices, 1);
			mulBlockIds[pos] = makeVec3i(i, j, k);
		}
	}
}

__global__ void permuteNnzPerRowKernel(int size, const int* srcRowPtr, const int* P, int* nnzPerRow)
{
	const int rowId = blockIdx.x * blockDim.x + threadIdx.x;
	if (rowId >= size)
		return;

	nnzPerRow[P[rowId]] = srcRowPtr[rowId + 1] - srcRowPtr[rowId];
}

__global__ void permuteColIndKernel(int size, const int* srcRowPtr, const int* srcColInd, const int* P,
	int* dstColInd, int* dstMap, int* nnzPerRow)
{
	const int rowId = blockIdx.x * blockDim.x + threadIdx.x;
	if (rowId >= size)
		return;

	const int i0 = srcRowPtr[rowId];
	const int i1 = srcRowPtr[rowId + 1];
	const int permRowId = P[rowId];
	for (int srck = i0; srck < i1; srck++)
	{
		const int dstk = nnzPerRow[permRowId]++;
		dstColInd[dstk] = P[srcColInd[srck]];
		dstMap[dstk] = srck;
	}
}

__global__ void schurComplementPostKernel(int cols, LxLBlockPtr invHll, Lx1BlockPtr bl, PxLBlockPtr Hpl,
	const int* HplColPtr, const int* HplRowInd, Px1BlockPtr xp, Lx1BlockPtr xl)
{
	const int colId = blockIdx.x * blockDim.x + threadIdx.x;
	if (colId >= cols)
		return;

	Scalar cl[LDIM];
	copy<LDIM>(bl.at(colId), cl);

	for (int i = HplColPtr[colId]; i < HplColPtr[colId + 1]; i++)
		MatTMulVec<3, 6, DEACCUM>(Hpl.at(i), xp.at(HplRowInd[i]), cl, 1);

	MatMulVec<3, 3>(invHll.at(colId), cl, xl.at(colId));
}

__global__ void updatePosesKernel(int size, Px1BlockPtr xp, Vec4d* qs, Vec3d* ts)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	Vec4d expq;
	Vec3d expt;
	updateExp(xp.at(i), expq, expt);
	updatePose(expq, expt, qs[i], ts[i]);
}

__global__ void updateLandmarksKernel(int size, Lx1BlockPtr xl, Vec3d* Xws)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	const Scalar* dXw = xl.at(i);
	Vec3d& Xw = Xws[i];
	Xw[0] += dXw[0];
	Xw[1] += dXw[1];
	//Xw[2] += dXw[2];
}

__global__ void computeScaleKernel(const Scalar* x, const Scalar* b, Scalar* scale, Scalar lambda, int size)
{
	const int sharedIdx = threadIdx.x;
	__shared__ Scalar cache[BLOCK_COMPUTE_SCALE];

	Scalar sum = 0;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x)
		sum += x[i] * (lambda * x[i] + b[i]);

	cache[sharedIdx] = sum;
	__syncthreads();

	for (int stride = BLOCK_COMPUTE_SCALE / 2; stride > 0; stride >>= 1)
	{
		if (sharedIdx < stride)
			cache[sharedIdx] += cache[sharedIdx + stride];
		__syncthreads();
	}

	if (sharedIdx == 0)
		atomicAdd(scale, cache[0]);
}

__global__ void convertBSRToCSRKernel(int size, const Scalar* src, Scalar* dst, const int* map)
{
	const int dstk = blockIdx.x * blockDim.x + threadIdx.x;
	if (dstk >= size)
		return;

	dst[dstk] = src[map[dstk]];
}

__global__ void nnzPerColKernel(const Vec3i* blockpos, int nblocks, int* nnzPerCol)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nblocks)
		return;

	const int colId = blockpos[i][1];
	atomicAdd(&nnzPerCol[colId], 1);
}

__global__ void setRowIndKernel(const Vec3i* blockpos, int nblocks, int* rowInd, int* indexPL)
{
	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if (k >= nblocks)
		return;

	const int rowId = blockpos[k][0];
	const int edgeId = blockpos[k][2];
	rowInd[k] = rowId;
	indexPL[edgeId] = k;
}

__global__ void solveDiagonalSystemKernel(int size, LxLBlockPtr Hll, Lx1BlockPtr bl, Lx1BlockPtr xl)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	solveSym3x3(Hll.at(i), bl.at(i), xl.at(i));
}

__global__ void solveDiagonalSystemKernel(int size, PxPBlockPtr Hpp, Px1BlockPtr bp, Px1BlockPtr xp)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;

	solveSym6x6(Hpp.at(i), bp.at(i), xp.at(i));
}

////////////////////////////////////////////////////////////////////////////////////
// Public functions
////////////////////////////////////////////////////////////////////////////////////

void waitForKernelCompletion()
{
	CUDA_CHECK(hipDeviceSynchronize());
}

void exclusiveScan(const int* src, int* dst, int size)
{
	auto ptrSrc = thrust::device_pointer_cast(src);
	auto ptrDst = thrust::device_pointer_cast(dst);
	thrust::exclusive_scan(ptrSrc, ptrSrc + size, ptrDst);
}

void buildHplStructure(GpuVec3i& blockpos, GpuHplBlockMat& Hpl, GpuVec1i& indexPL, GpuVec1i& nnzPerCol)
{
	const int nblocks = Hpl.nnz();
	const int block = 1024;
	const int grid = divUp(nblocks, block);
	int* colPtr = Hpl.outerIndices();
	int* rowInd = Hpl.innerIndices();

	auto ptrBlockPos = thrust::device_pointer_cast(blockpos.data());
	thrust::sort(ptrBlockPos, ptrBlockPos + nblocks, LessColId());

	CUDA_CHECK(hipMemset(nnzPerCol, 0, sizeof(int) * (Hpl.cols() + 1)));
	nnzPerColKernel<<<grid, block>>>(blockpos, nblocks, nnzPerCol);
	exclusiveScan(nnzPerCol, colPtr, Hpl.cols() + 1);
	setRowIndKernel<<<grid, block>>>(blockpos, nblocks, rowInd, indexPL);
}

void findHschureMulBlockIndices(const GpuHplBlockMat& Hpl, const GpuHscBlockMat& Hsc,
	GpuVec3i& mulBlockIds)
{
	const int block = 1024;
	const int grid = divUp(Hpl.cols(), block);

	DeviceBuffer<int> nindices(1);
	nindices.fillZero();

	findHschureMulBlockIndicesKernel<<<grid, block>>>(Hpl.cols(), Hpl.outerIndices(), Hpl.innerIndices(),
		Hsc.outerIndices(), Hsc.innerIndices(), mulBlockIds, nindices);
	CUDA_CHECK(hipGetLastError());

	auto ptrSrc = thrust::device_pointer_cast(mulBlockIds.data());
	thrust::sort(ptrSrc, ptrSrc + mulBlockIds.size(), LessRowId());
}

template <int MDIM, int RK_TYPE = 0>
Scalar computeActiveErrors_(const GpuVec4d& qs, const GpuVec3d& ts, const GpuVec5d& cameras, const GpuVec3d& Xws,
	const GpuVecAny& _measurements, const GpuVec1d& omegas, const GpuVec2i& edge2PL, Scalar robustDelta,
	const GpuVecAny& _errors, GpuVec3d& Xcs, Scalar* chi)
{
	const auto& measurements = _measurements.getCRef<Vecxd<MDIM>>();
	auto& errors = _errors.getRef<Vecxd<MDIM>>();
	const RobustKernelFunc<RK_TYPE> robustKernel(robustDelta);

	const int nedges = measurements.ssize();
	const int block = BLOCK_ACTIVE_ERRORS;
	const int grid = 16;

	if (nedges <= 0)
		return 0;

	CUDA_CHECK(hipMemset(chi, 0, sizeof(Scalar)));
	computeActiveErrorsKernel<MDIM, RK_TYPE><<<grid, block>>>(nedges, qs, ts, cameras, Xws, measurements, omegas,
		edge2PL, robustKernel, errors, Xcs, chi);
	CUDA_CHECK(hipGetLastError());

	Scalar h_chi = 0;
	CUDA_CHECK(hipMemcpy(&h_chi, chi, sizeof(Scalar), hipMemcpyDeviceToHost));

	return h_chi;
}

using ComputeActiveErrorsFunc = Scalar(*)(const GpuVec4d&, const GpuVec3d&, const GpuVec5d&, const GpuVec3d&,
	const GpuVecAny&, const GpuVec1d&, const GpuVec2i&, Scalar, const GpuVecAny&, GpuVec3d&, Scalar*);

static ComputeActiveErrorsFunc computeActiveErrorsFuncs[6] =
{
	computeActiveErrors_<2, 0>,
	computeActiveErrors_<2, 1>,
	computeActiveErrors_<2, 2>,
	computeActiveErrors_<3, 0>,
	computeActiveErrors_<3, 1>,
	computeActiveErrors_<3, 2>
};

Scalar computeActiveErrors(const GpuVec4d& qs, const GpuVec3d& ts, const GpuVec5d& cameras, const GpuVec3d& Xws,
	const GpuVec2d& measurements, const GpuVec1d& omegas, const GpuVec2i& edge2PL, const RobustKernel& kernel,
	GpuVec2d& errors, GpuVec3d& Xcs, Scalar* chi)
{
	auto func = computeActiveErrorsFuncs[0 + kernel.type];
	return func(qs, ts, cameras, Xws, measurements, omegas, edge2PL, kernel.delta, errors, Xcs, chi);
}

Scalar computeActiveErrors(const GpuVec4d& qs, const GpuVec3d& ts, const GpuVec5d& cameras, const GpuVec3d& Xws,
	const GpuVec3d& measurements, const GpuVec1d& omegas, const GpuVec2i& edge2PL, const RobustKernel& kernel,
	GpuVec3d& errors, GpuVec3d& Xcs, Scalar* chi)
{
	auto func = computeActiveErrorsFuncs[3 + kernel.type];
	return func(qs, ts, cameras, Xws, measurements, omegas, edge2PL, kernel.delta, errors, Xcs, chi);
}

template <int MDIM, int RK_TYPE = 0>
void constructQuadraticForm_(const GpuVec3d& Xcs, const GpuVec4d& qs, const GpuVec5d& cameras, const GpuVecAny& _errors,
	const GpuVec1d& omegas, const GpuVec2i& edge2PL, const GpuVec1i& edge2Hpl, const GpuVec1b& flags, Scalar robustDelta,
	GpuPxPBlockVec& Hpp, GpuPx1BlockVec& bp, GpuLxLBlockVec& Hll, GpuLx1BlockVec& bl, GpuHplBlockMat& Hpl)
{
	const auto& errors = _errors.getRef<Vecxd<MDIM>>();
	const RobustKernelFunc<RK_TYPE> robustKernel(robustDelta);

	const int nedges = errors.ssize();
	const int block = 512;
	const int grid = divUp(nedges, block);

	if (nedges <= 0)
		return;

	constructQuadraticFormKernel<MDIM, RK_TYPE><<<grid, block>>>(nedges, Xcs, qs, cameras, errors, omegas,
		edge2PL, edge2Hpl, flags, robustKernel, Hpp, bp, Hll, bl, Hpl);
	CUDA_CHECK(hipGetLastError());
}

using ConstructQuadraticFormFunc = void(*)(const GpuVec3d&, const GpuVec4d&, const GpuVec5d&, const GpuVecAny&,
	const GpuVec1d&, const GpuVec2i&, const GpuVec1i&, const GpuVec1b&, Scalar,
	GpuPxPBlockVec&, GpuPx1BlockVec&, GpuLxLBlockVec&, GpuLx1BlockVec&, GpuHplBlockMat&);

static ConstructQuadraticFormFunc constructQuadraticFormFuncs[6] =
{
	constructQuadraticForm_<2, 0>,
	constructQuadraticForm_<2, 1>,
	constructQuadraticForm_<2, 2>,
	constructQuadraticForm_<3, 0>,
	constructQuadraticForm_<3, 1>,
	constructQuadraticForm_<3, 2>
};

void constructQuadraticForm(const GpuVec3d& Xcs, const GpuVec4d& qs, const GpuVec5d& cameras, const GpuVec2d& errors,
	const GpuVec1d& omegas, const GpuVec2i& edge2PL, const GpuVec1i& edge2Hpl, const GpuVec1b& flags, const RobustKernel& kernel,
	GpuPxPBlockVec& Hpp, GpuPx1BlockVec& bp, GpuLxLBlockVec& Hll, GpuLx1BlockVec& bl, GpuHplBlockMat& Hpl)
{
	auto func = constructQuadraticFormFuncs[0 + kernel.type];
	func(Xcs, qs, cameras, errors, omegas, edge2PL, edge2Hpl, flags, kernel.delta, Hpp, bp, Hll, bl, Hpl);
}

void constructQuadraticForm(const GpuVec3d& Xcs, const GpuVec4d& qs, const GpuVec5d& cameras, const GpuVec3d& errors,
	const GpuVec1d& omegas, const GpuVec2i& edge2PL, const GpuVec1i& edge2Hpl, const GpuVec1b& flags, const RobustKernel& kernel,
	GpuPxPBlockVec& Hpp, GpuPx1BlockVec& bp, GpuLxLBlockVec& Hll, GpuLx1BlockVec& bl, GpuHplBlockMat& Hpl)
{
	auto func = constructQuadraticFormFuncs[3 + kernel.type];
	func(Xcs, qs, cameras, errors, omegas, edge2PL, edge2Hpl, flags, kernel.delta, Hpp, bp, Hll, bl, Hpl);
}

template <int MDIM>
void computeChiSquares_(const GpuVec4d& qs, const GpuVec3d& ts, const GpuVec5d& cameras, const GpuVec3d& Xws,
	const GpuVecAny& _measurements, const GpuVec1d& omegas, const GpuVec2i& edge2PL, GpuVec1d& chiSqs)
{
	using Vecmd = Vecxd<MDIM>;

	const GpuVec<Vecmd>& measurements = _measurements.getCRef<Vecmd>();

	const int nedges = measurements.ssize();
	const int block = 512;
	const int grid = divUp(nedges, block);

	if (nedges <= 0)
		return;

	computeChiSquaresKernel<MDIM><<<grid, block>>>(nedges, qs, ts, cameras, Xws, measurements, omegas, edge2PL, chiSqs);
	CUDA_CHECK(hipGetLastError());
}

void computeChiSquares(const GpuVec4d& qs, const GpuVec3d& ts, const GpuVec5d& cameras, const GpuVec3d& Xws,
	const GpuVec2d& measurements, const GpuVec1d& omegas, const GpuVec2i& edge2PL, GpuVec1d& chiSqs)
{
	computeChiSquares_<2>(qs, ts, cameras, Xws, measurements, omegas, edge2PL, chiSqs);
}

void computeChiSquares(const GpuVec4d& qs, const GpuVec3d& ts, const GpuVec5d& cameras, const GpuVec3d& Xws,
	const GpuVec3d& measurements, const GpuVec1d& omegas, const GpuVec2i& edge2PL, GpuVec1d& chiSqs)
{
	computeChiSquares_<3>(qs, ts, cameras, Xws, measurements, omegas, edge2PL, chiSqs);
}

template <typename T, int DIM>
Scalar maxDiagonal_(const DeviceBlockVector<T, DIM, DIM>& D, Scalar* maxD)
{
	if (!D.size())
		return 0;

	constexpr int block = BLOCK_MAX_DIAGONAL;
	constexpr int grid = 4;
	const int size = D.size() * DIM;

	maxDiagonalKernel<DIM><<<grid, block>>>(size, D.values(), maxD);
	CUDA_CHECK(hipGetLastError());

	Scalar tmpMax[grid];
	CUDA_CHECK(hipMemcpy(tmpMax, maxD, sizeof(Scalar) * grid, hipMemcpyDeviceToHost));

	Scalar maxv = 0;
	for (int i = 0; i < grid; i++)
		maxv = std::max(maxv, tmpMax[i]);

	return maxv;
}

Scalar maxDiagonal(const GpuPxPBlockVec& Hpp, Scalar* maxD)
{
	return maxDiagonal_(Hpp, maxD);
}

Scalar maxDiagonal(const GpuLxLBlockVec& Hll, Scalar* maxD)
{
	return maxDiagonal_(Hll, maxD);
}

template <typename T, int DIM>
void addLambda_(DeviceBlockVector<T, DIM, DIM>& D, Scalar lambda, DeviceBlockVector<T, DIM, 1>& backup)
{
	if (!D.size())
		return;

	const int size = D.size() * DIM;
	const int block = 1024;
	const int grid = divUp(size, block);
	addLambdaKernel<DIM><<<grid, block>>>(size, D.values(), lambda, backup.values());
	CUDA_CHECK(hipGetLastError());
}

void addLambda(GpuPxPBlockVec& Hpp, Scalar lambda, GpuPx1BlockVec& backup)
{
	addLambda_(Hpp, lambda, backup);
}

void addLambda(GpuLxLBlockVec& Hll, Scalar lambda, GpuLx1BlockVec& backup)
{
	addLambda_(Hll, lambda, backup);
}

template <typename T, int DIM>
void restoreDiagonal_(DeviceBlockVector<T, DIM, DIM>& D, const DeviceBlockVector<T, DIM, 1>& backup)
{
	if (!D.size())
		return;

	const int size = D.size() * DIM;
	const int block = 1024;
	const int grid = divUp(size, block);
	restoreDiagonalKernel<DIM><<<grid, block>>>(size, D.values(), backup.values());
	CUDA_CHECK(hipGetLastError());
}

void restoreDiagonal(GpuPxPBlockVec& Hpp, const GpuPx1BlockVec& backup)
{
	restoreDiagonal_(Hpp, backup);
}

void restoreDiagonal(GpuLxLBlockVec& Hll, const GpuLx1BlockVec& backup)
{
	restoreDiagonal_(Hll, backup);
}

void computeBschure(const GpuPx1BlockVec& bp, const GpuHplBlockMat& Hpl, const GpuLxLBlockVec& Hll,
	const GpuLx1BlockVec& bl, GpuPx1BlockVec& bsc, GpuLxLBlockVec& invHll, GpuPxLBlockVec& Hpl_invHll)
{
	const int cols = Hll.size();
	const int block = 256;
	const int grid = divUp(cols, block);

	bp.copyTo(bsc);
	computeBschureKernel<<<grid, block>>>(cols, Hll, invHll, bl, Hpl, Hpl.outerIndices(), Hpl.innerIndices(),
		bsc, Hpl_invHll);
	CUDA_CHECK(hipGetLastError());
}

void computeHschure(const GpuPxPBlockVec& Hpp, const GpuPxLBlockVec& Hpl_invHll,
	const GpuHplBlockMat& Hpl, const GpuVec3i& mulBlockIds, GpuHscBlockMat& Hsc)
{
	const int nmulBlocks = mulBlockIds.ssize();
	const int block = 256;
	const int grid1 = divUp(Hsc.rows(), block);
	const int grid2 = divUp(nmulBlocks, block);

	Hsc.fillZero();
	initializeHschurKernel<<<grid1, block>>>(Hsc.rows(), Hpp, Hsc, Hsc.outerIndices());
	computeHschureKernel<<<grid2, block>>>(nmulBlocks, mulBlockIds, Hpl_invHll, Hpl, Hsc);
	CUDA_CHECK(hipGetLastError());
}

void convertHschureBSRToCSR(const GpuHscBlockMat& HscBSR, const GpuVec1i& BSR2CSR, GpuVec1d& HscCSR)
{
	const int size = HscCSR.ssize();
	const int block = 1024;
	const int grid = divUp(size, block);
	convertBSRToCSRKernel<<<grid, block>>>(size, HscBSR.values(), HscCSR, BSR2CSR);
}

void twistCSR(int size, int nnz, const int* srcRowPtr, const int* srcColInd, const int* P,
	int* dstRowPtr, int* dstColInd, int* dstMap, int* nnzPerRow)
{
	const int block = 512;
	const int grid = divUp(size, block);

	permuteNnzPerRowKernel<<<grid, block>>>(size, srcRowPtr, P, nnzPerRow);
	exclusiveScan(nnzPerRow, dstRowPtr, size + 1);
	CUDA_CHECK(hipMemcpy(nnzPerRow, dstRowPtr, sizeof(int) * (size + 1), hipMemcpyDeviceToDevice));
	permuteColIndKernel<<<grid, block>>>(size, srcRowPtr, srcColInd, P, dstColInd, dstMap, nnzPerRow);
}

void permute(int size, const Scalar* src, Scalar* dst, const int* P)
{
	auto ptrSrc = thrust::device_pointer_cast(src);
	auto ptrDst = thrust::device_pointer_cast(dst);
	auto ptrMap = thrust::device_pointer_cast(P);
	thrust::gather(ptrMap, ptrMap + size, ptrSrc, ptrDst);
}

void schurComplementPost(const GpuLxLBlockVec& invHll, const GpuLx1BlockVec& bl,
	const GpuHplBlockMat& Hpl, const GpuPx1BlockVec& xp, GpuLx1BlockVec& xl)
{
	const int block = 1024;
	const int grid = divUp(Hpl.cols(), block);

	schurComplementPostKernel<<<grid, block>>>(Hpl.cols(), invHll, bl, Hpl,
		Hpl.outerIndices(), Hpl.innerIndices(),xp, xl);
	CUDA_CHECK(hipGetLastError());
}

void updatePoses(const GpuPx1BlockVec& xp, GpuVec4d& qs, GpuVec3d& ts)
{
	if (!xp.size())
		return;

	const int block = 256;
	const int grid = divUp(xp.size(), block);
	updatePosesKernel<<<grid, block>>>(xp.size(), xp, qs, ts);
	CUDA_CHECK(hipGetLastError());
}

void updateLandmarks(const GpuLx1BlockVec& xl, GpuVec3d& Xws)
{
	if (!xl.size())
		return;

	const int block = 1024;
	const int grid = divUp(xl.size(), block);
	updateLandmarksKernel<<<grid, block>>>(xl.size(), xl, Xws);
	CUDA_CHECK(hipGetLastError());
}

void computeScale(const GpuVec1d& x, const GpuVec1d& b, Scalar* scale, Scalar lambda)
{
	const int block = BLOCK_COMPUTE_SCALE;
	const int grid = 4;

	CUDA_CHECK(hipMemset(scale, 0, sizeof(Scalar)));
	computeScaleKernel<<<grid, block>>>(x, b, scale, lambda, x.ssize());
	CUDA_CHECK(hipGetLastError());
}

void solveDiagonalSystem(const GpuLxLBlockVec& Hll, GpuLx1BlockVec& bl, GpuLx1BlockVec& xl)
{
	const int size = Hll.size();
	const int block = 1024;
	const int grid = divUp(size, block);
	solveDiagonalSystemKernel<<<grid, block>>>(size, Hll, bl, xl);
	CUDA_CHECK(hipGetLastError());
}

void solveDiagonalSystem(const GpuPxPBlockVec& Hpp, GpuPx1BlockVec& bp, GpuPx1BlockVec& xp)
{
	const int size = Hpp.size();
	const int block = 512;
	const int grid = divUp(size, block);
	solveDiagonalSystemKernel<<<grid, block>>>(size, Hpp, bp, xp);
	CUDA_CHECK(hipGetLastError());
}

} // namespace gpu
} // namespace cuba
